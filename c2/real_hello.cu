
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
    printf("hello world from the gpu\n");
}

int main(int argc, char const *argv[])
{   
    // 三括号中的第一个数字可以看作线程块的个数，第二个数字可以看作每个线程块中的线程数
    hello_from_gpu<<<1, 1>>>(); 

    // 一个cuda的运行时API函数，作用是同步主机与设备（促使缓冲区刷新）
    hipDeviceSynchronize();
    return 0;
}
