#include "hip/hip_runtime.h"
#include "error.cuh"
#include <math.h>
#include <stdio.h>

const double EPSILON = 1.0e-15;
const double a = 1.23;
const double b = 2.34;
const double c = 3.57;
void __global__ add(const double *x, const double *y, double *z, const int N);
void check(const double *z, const int N);

int main(void)
{
    const int N = 100000000;
    const int M = sizeof(double) * N;
    double *h_x = (double*) malloc(M);
    double *h_y = (double*) malloc(M);
    double *h_z = (double*) malloc(M);

    for (int n = 0; n < N; ++n)
    {
        h_x[n] = a;
        h_y[n] = b;
    }

    double *d_x, *d_y, *d_z;
    CHECK(hipMalloc((void **)&d_x, M));
    CHECK(hipMalloc((void **)&d_y, M));
    CHECK(hipMalloc((void **)&d_z, M));
    CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice));

    const int block_size = 1280; // 线程块大小的最大值是1024（这对从开普勒到图灵的所有架构都成立）
    const int grid_size = (N + block_size - 1) / block_size;
    add<<<grid_size, block_size>>>(d_x, d_y, d_z, N);

    /*
        第一个语句的作用是捕捉第二个语句之前的最后一个错误
        第二个语句的作用是同步主机与设备
            之所以要同步主机与设备，是因为核函数的调用是异步的，即主机发出调用核函数的命令后会立即执行后面的语句，不会等待核函数执行完毕
            这样设置之后，所有核函数的调用都将不再是异步的，而是同步的。也就是说，主机调用一个核函数之后，必须等待核函数执行完毕，才能往下走。
            这样的设置一般来说仅适用于调试程序，因为它会影响程序的性能
    */
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost));
    check(h_z, N);

    free(h_x);
    free(h_y);
    free(h_z);
    CHECK(hipFree(d_x));
    CHECK(hipFree(d_y));
    CHECK(hipFree(d_z));
    return 0;
}

void __global__ add(const double *x, const double *y, double *z, const int N)
{
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N)
    {
        z[n] = x[n] + y[n];
    }
}

void check(const double *z, const int N)
{
    bool has_error = false;
    for (int n = 0; n < N; ++n)
    {
        if (fabs(z[n] - c) > EPSILON)
        {
            has_error = true;
        }
    }
    printf("%s\n", has_error ? "Has errors" : "No errors");
}
